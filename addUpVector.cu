#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void addUpVector(int* input, int N) {
	int threadIndex = threadIdx.x + threadIdx.x;

	for (int gait=1; gait<N; gait+=gait) {
		if(threadIndex % (2*gait) == 0)
			input[threadIndex]+= input[threadIndex + gait];
		else
			break;
	}
	__syncthreads();

}

int main(){

	int threads = 501;
	int length = threads * 2;

	int host_v[length];
	for (int i = 0; i < length; i++) {
		host_v[i] = 1;
	}
	
	int* device_v;
	hipMalloc((void**)&device_v, length*sizeof(int));
	hipMemcpy(device_v, host_v, length*sizeof(int),hipMemcpyHostToDevice);

	dim3 blockSize(threads);
	dim3 gridSize(1);

	addUpVector<<<gridSize,blockSize>>>(device_v, length);

	hipMemcpy(host_v, device_v, length*sizeof(int),hipMemcpyDeviceToHost);

	printf("%d", host_v[0]);

	hipFree(device_v);

	return 0;

}


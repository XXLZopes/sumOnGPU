#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void addUpVector(int* input, int N) {
	int threadIndex = threadIdx.x;
	for (int gait=1; gait<N; gait*=2) {
		if(threadIndex % (2*gait) == 0 && gait < N)
			input[threadIndex]+= input[threadIndex + gait];
		__syncthreads();
	}
}

int main(){

	int length = 1000;

	int host_v[length];
	for (int i = 0; i < length; i++) {
		host_v[i] = 1;
	}
	
	int* device_v;
	hipMalloc((void**)&device_v, length*sizeof(int));
	hipMemcpy(device_v, host_v, length*sizeof(int),hipMemcpyHostToDevice);

	dim3 blockSize(length);
	dim3 gridSize(1);

	addUpVector<<<gridSize,blockSize>>>(device_v, length);

	hipMemcpy(host_v, device_v, length*sizeof(int),hipMemcpyDeviceToHost);

	printf("%d \n\n\n", host_v[0]);

	hipFree(device_v);

	return 0;

}
